#include "hip/hip_runtime.h"
#include "CUDA_functions.cuh"
#include "CUDA_exception.h"
#include "CUDA_malloc_exception.h"
#include "CUDA_memcpy_exception.h"
#include "CUDA_device_exception.h"
#include "CUDA_kernel_exception.h"
#include "CUDA_synchronize_exception.h"
#include "CUDA_error.h"

int answersNumber;
int categoriesNumber;
int atribsNumber;

/**
* Funkcja wykonywana na karcie graficznej - kazdy watek sprawdza czy jego atrybut z atribsValues to ten sam co w query. Jesli tak, przepisuje do
* tablicy wynikowej prawdopodobiestwa dla kazdej jego odpowiedzi
* @param query - zapytanie uzytkownika w postacie zlepionych stringow
* @param atribsValues - tablica wszystkich atrybutow
* @param possibilities - tablica wszystkich prawdopodobienstw
* @param queryPrefix - tablica sum prefiksowych dlugosci slow w query
* @param atribsPrefix - j.k. dla atribsValues
* @param answersNumber - liczba mozliwych odpowiedzi
* @param categoriesNumber - liczba kategorii
* @param atribsNumber - liczba wszystkich atrybutow
* @param resultPossibilities - tablica prawdopodobienstw atrybutow z zapytania dla wszystkich mozliwych odpowiedzi
*/
__global__ void searchWithCuda(double *resultPossibilities, char *query, char *atribsValues, double *possibilities, int *queryPrefix, int *atribsPrefix, int *answersNumber, int *categoriesNumber, int *atribsNumber)
{
	int category_id = blockIdx.x;	// categories
	int atrib_id = blockIdx.y;	// atribs

								// znajdz poczatek lancucha znakow atrybutu w zapytaniu i w atribsValue
	char *queryAtrib = query + queryPrefix[category_id];
	int queryAtribLength = queryPrefix[category_id + 1] - queryPrefix[category_id];

	char *currAtrib = atribsValues + atribsPrefix[atrib_id];
	int currAtribLength = atribsPrefix[atrib_id + 1] - atribsPrefix[atrib_id];

	if (queryAtribLength == currAtribLength)
	{
		bool equal = true;
		for (int i = 0; i < queryAtribLength; ++i)
		{
			if (queryAtrib[i] != currAtrib[i])
			{
				equal = false;
				break;
			}
		}
		if (equal)	// przypisz odpowiednie prawdopodobienstwa
		{
			for (int i = 0; i < *answersNumber; ++i)
			{
				resultPossibilities[*categoriesNumber*i + category_id] = possibilities[*atribsNumber*i + atrib_id];	// na razie tylko dla jednej odpowiedzi
			}
		}
	}
}

void setDevice()
{
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
		throw CUDA_device_exception();
}

void getLastError()
{
	hipError_t code;
	if ((code = hipGetLastError()) != hipSuccess)
		throw CUDA_error(code);
}

void deviceSynchronize()
{
	if (hipDeviceSynchronize() != hipSuccess)
		throw CUDA_synchronize_exception();
}

/**
* Funkcja wywolujaca funkcje dzialajaca na GPU. Po uzyskaniu wynikowej tablicy prawdopodobienstw wymnaza prawdopodobienstwa dla kazdej odpowiedzi i
* wybiera najlepiej dopasowana
* @param query - zapytanie uzytkownika w postacie zlepionych stringow
* @param atribsValues - tablica wszystkich atrybutow
* @param possibilities - tablica wszystkich prawdopodobienstw
* @param queryPrefix - tablica sum prefiksowych dlugosci slow w query
* @param atribsPrefix - j.k. dla atribsValues
* @param answersNumber - liczba mozliwych odpowiedzi
* @param categoriesNumber - liczba kategorii
* @param atribsNumber - liczba wszystkich atrybutow
* @return indeks najlepiej dopasowanej odpowiedzi
*/
__host__ int findAnswer(char *query, char *atribsValues, double *possibilities, int *queryPrefix, int *atribsPrefix, int answersNumber, int categoriesNumber, int atribsNumber)
{
	int queryCharNumber = queryPrefix[categoriesNumber];
	int atribsCharNumber = atribsPrefix[atribsNumber];

	double* resultPossibilities = new double[answersNumber*categoriesNumber];

	double *dev_resultPossibilities = 0;
	char *dev_query = 0;
	char *dev_atribsValues = 0;
	double *dev_possibilities = 0;
	int *dev_queryPrefix = 0;
	int *dev_atribsPrefix = 0;
	int *dev_answersNumber = 0;
	int *dev_categoriesNumber = 0;
	int *dev_atribsNumber = 0;

	try
	{
		setDevice();

		if (hipMalloc((void**)&dev_resultPossibilities, answersNumber * categoriesNumber * sizeof(double)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_query, queryCharNumber * sizeof(char)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_atribsValues, atribsCharNumber * sizeof(char)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_possibilities, answersNumber * atribsNumber * sizeof(double)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_queryPrefix, (categoriesNumber + 1) * sizeof(int)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_atribsPrefix, (atribsNumber + 1) * sizeof(int)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_answersNumber, sizeof(int)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_categoriesNumber, sizeof(int)) != hipSuccess)
			throw CUDA_malloc_exception();
		if (hipMalloc((void**)&dev_atribsNumber, sizeof(int)) != hipSuccess)
			throw CUDA_malloc_exception();

		if (hipMemcpy(dev_query, query, queryCharNumber * sizeof(char), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();
		if (hipMemcpy(dev_atribsValues, atribsValues, atribsCharNumber * sizeof(char), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();
		if (hipMemcpy(dev_possibilities, possibilities, answersNumber * atribsNumber * sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();
		if (hipMemcpy(dev_queryPrefix, queryPrefix, (categoriesNumber + 1) * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();
		if (hipMemcpy(dev_atribsPrefix, atribsPrefix, (atribsNumber + 1) * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();
		if (hipMemcpy(dev_answersNumber, &answersNumber, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();
		if (hipMemcpy(dev_categoriesNumber, &categoriesNumber, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();
		if (hipMemcpy(dev_atribsNumber, &atribsNumber, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
			throw CUDA_memcpy_exception();


		dim3 dimBlock(categoriesNumber, atribsNumber, 1); // x, y, z
		searchWithCuda << <dimBlock, 1 >> >(dev_resultPossibilities, dev_query, dev_atribsValues, dev_possibilities, dev_queryPrefix, dev_atribsPrefix, dev_answersNumber, dev_categoriesNumber, dev_atribsNumber);

		getLastError();
		deviceSynchronize();


		if (hipMemcpy(resultPossibilities, dev_resultPossibilities, answersNumber * categoriesNumber * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess)
			throw CUDA_memcpy_exception();
	}
	catch (exception& e)
	{
		printf("%s", e.what());
		hipFree(dev_atribsPrefix);
		hipFree(dev_atribsValues);
		hipFree(dev_possibilities);
		hipFree(dev_query);
		hipFree(dev_queryPrefix);
		hipFree(dev_resultPossibilities);
		hipFree(dev_answersNumber);
		hipFree(dev_categoriesNumber);
		hipFree(dev_atribsNumber);
		delete[] resultPossibilities;

		return -1;
	}

	hipFree(dev_atribsPrefix);
	hipFree(dev_atribsValues);
	hipFree(dev_possibilities);
	hipFree(dev_query);
	hipFree(dev_queryPrefix);
	hipFree(dev_resultPossibilities);
	hipFree(dev_answersNumber);
	hipFree(dev_categoriesNumber);
	hipFree(dev_atribsNumber);


	double *answersPos = new double[answersNumber];
	double max = 0.0;
	int maxId = 0;
	for (int i = 0; i < answersNumber; ++i)
	{
		answersPos[i] = 1;
		for (int j = 0; j < categoriesNumber; ++j)
			answersPos[i] *= resultPossibilities[i*categoriesNumber + j];
		if (answersPos[i] > max)
		{
			maxId = i;
			max = answersPos[i];
		}
	}


	delete[] resultPossibilities;
	delete[] answersPos;
	return maxId;
}